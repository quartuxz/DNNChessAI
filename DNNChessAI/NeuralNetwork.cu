#include "hip/hip_runtime.h"
#include "NeuralNetwork.cuh"
#include <random>
#include <math.h>
#include <stdexcept>
#include <chrono>
#include <iostream>


#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>


std::default_random_engine NNInitialization::engine = std::default_random_engine(static_cast<long unsigned int>(time(0)));


#define THREADS_PER_BLOCK 512



void NeuralNetwork::m_initializeGpuMem()
{

	// Choose which GPU to run on, change this on a multi-GPU system.
	hipError_t cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
	}

	//initialize the GPU memory
	for (size_t i = 0; i < m_hiddenLayers.size() + 1; i++) {
		Layer* layer;

		if (i < m_hiddenLayers.size()) {
			layer = &m_hiddenLayers[i];
		}
		else {
			layer = &m_outputLayer;
		}
		size_t allSynapsesSize = layer->prevSize * layer->size;
		GPUDataLayer data = std::make_tuple<float*, float*, unsigned int*>(0, 0, 0);

		cudaStatus = hipMalloc((void**)&std::get<0>(data), allSynapsesSize * sizeof(float));
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMalloc failed!");
		}

		cudaStatus = hipMalloc((void**)&std::get<1>(data), layer->size * sizeof(float));
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMalloc failed!");
		}

		cudaStatus = hipMalloc((void**)&std::get<2>(data), 3 * sizeof(unsigned int));
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMalloc failed!");
		}
		m_GPULayers.push_back(data);
	}
}

//we build up the neural network
NeuralNetwork::NeuralNetwork(Topology top, NNInitialization init, LearningSchedule ls, NNActivation act):
	m_top(top),
	m_act(act),
	m_learningSched(ls),
	m_outputLayer(init,top.back(),top[top.size()-2],true)
{
	//we create the layers
	for (size_t i = 0; i < top.size()-2; i++)
	{
		m_hiddenLayers.push_back(Layer(init, top[i + 1], top[i]));
	}

	m_initializeGpuMem();
	//TODO: optimize this to not run during deserializing 
	m_updateGPUMem();
}


Topology getTopology(const std::string& str) {
	std::string currentNumber;

	Topology retval;

	for (char current : str) {
		if (current == '\n') {
			break;
		}
		if (current == ' ') {
			retval.push_back(std::atoi(currentNumber.c_str()));
			std::cout << currentNumber << std::endl;
			currentNumber = "";
			continue;
		}
		else {
			currentNumber.push_back(current);
		}
		
	}
	return retval;
}

NeuralNetwork::NeuralNetwork(const std::string& str) :
	NeuralNetwork(getTopology(str),NNInitialization(),LearningSchedule())
{
	std::string currentNumber;

	std::vector<Layer*> allLayers;

	for (auto& layer : m_hiddenLayers) {
		allLayers.push_back(&layer);
	}
	allLayers.push_back(&m_outputLayer);
	
	//line of the string being read
	unsigned int line = 0;

	//orienting indexes for creating the NN with the corresponding values as read.
	unsigned int layerN = 0, neuronN = 0, synapseN = 0;
	bool isBias = true;
	for (char current : str) {
		//the first line was already read for topology information.
		if (current == '\n') {
			if (line >= 1) {
				break;
			}
			line++;
			continue;
		}
		if (line < 1) {
			continue;
		}
		

		if (current == ' ') {
			float parsedNumber = std::atof(currentNumber.c_str());
			if (isBias) {
				allLayers[layerN]->biases[neuronN] = parsedNumber;
				isBias = false;
			}
			else {
				allLayers[layerN]->weights[neuronN][synapseN] = parsedNumber;
				currentNumber = "";
				synapseN++;
			}
			continue;
		}
		else if (current == '/') {
			isBias = true;
			neuronN++;
			synapseN = 0;
		}
		else if (current == ',') {
			layerN++;
			neuronN = 0;
		}
		else {
			currentNumber.push_back(current);
		}

	}
	m_updateGPUMem();
}



void NeuralNetwork::addRandomWeights()
{

	for (size_t i = 0; i < m_hiddenLayers.size() + 1; i++)
	{
		Layer* layer;
		if (i < m_hiddenLayers.size()) {
			layer = &m_hiddenLayers[i];
		}
		else {
			layer = &m_outputLayer;
		}

		layer->randomizeParameters(m_learningSched);
	}
	m_updateGPUMem();
}

NeuralNetwork::NeuralNetwork(const NeuralNetwork &other):
	m_outputLayer(other.m_outputLayer),
	m_act(other.m_act),
	m_hiddenLayers(other.m_hiddenLayers),
	m_learningSched(other.m_learningSched),
	m_top(other.m_top),
	m_batchN(other.m_batchN)
{
	m_initializeGpuMem();
	m_updateGPUMem();
}

NeuralNetwork& NeuralNetwork::operator=(NeuralNetwork other)
{
	swap(*this,other);
	return *this;
}

__global__ void calculateZErrorKernel(float *thisZError, float *thisBiasAcc, const float *thisZ,const float *thisA, const float *nextZError,  const float *weightMatrix, const unsigned int* extraParamsThisLayer, const unsigned int *extraParamsNextLayer)
{
	int neuron = threadIdx.x + blockDim.x * blockIdx.x;
	if (neuron < extraParamsThisLayer[2]) {
		const unsigned int &matrixWidth = extraParamsThisLayer[2];
		const unsigned int &matrixHeight = extraParamsNextLayer[2];
		float errorWeightSum = 0;
		for (size_t i = 0; i < matrixHeight; i++) {
			errorWeightSum += weightMatrix[neuron + matrixWidth * i]*nextZError[i];
		}
		float deriv = 0;
		if (extraParamsThisLayer[0] == 2) {

			deriv = thisA[neuron] * (1 - thisA[neuron]);
		}
		else if (extraParamsThisLayer[0] == 3) {
			float sigmoid = 1 / (1 + exp(-thisZ[neuron]));
			deriv = thisA[neuron]+sigmoid*(1-thisA[neuron]);
		}


		thisZError[neuron] = errorWeightSum * deriv;

		thisBiasAcc[neuron] += thisZError[neuron];
	}
}

__global__ void calculateWeightGradientsKernel(float *weightMatrixAcc, const float *thisZError, const float *prevA, const unsigned int *extraParams) {
	int neuron = threadIdx.x + blockDim.x * blockIdx.x;
	if (neuron < extraParams[2]) {
		for (size_t i = 0; i < extraParams[1]; i++) {
			weightMatrixAcc[i + neuron * extraParams[1]] += prevA[i]*thisZError[neuron];
		}
	}
}

//TODO: implement adam optimizer.
__global__ void addAverageWeightsAndBiasesKernel(float *weights, const float *weightsAcc, float *biases, const float *biasesAcc,const float* instancesAndLearningRate,const unsigned int* extraParams) {
	int neuron = threadIdx.x + blockDim.x * blockIdx.x;
	if (neuron < extraParams[2]) {
		for (size_t i = 0; i < extraParams[1]; i++) {
			weights[i + neuron * extraParams[1]] -= instancesAndLearningRate[1]*(weightsAcc[i + neuron * extraParams[1]]/instancesAndLearningRate[0]);
		}
		biases[neuron] -= instancesAndLearningRate[1] * (biases[neuron] / instancesAndLearningRate[0]);
	}
}


void NeuralNetwork::backpropagateGPU(std::vector<std::vector<float>>& dCost_dOutput_forInstances)
{
	m_instancesInBatch = 0;
	std::vector<float*> weightAcc;
	std::vector<float*> biasAcc;

	hipError_t cudaStatus = hipSetDevice(0);
	for (size_t i = 0; i < m_hiddenLayers.size() + 1; i++) {
		const Layer* layer = (i < m_hiddenLayers.size() ? &m_hiddenLayers[i] : &m_outputLayer);

		size_t allWeightsSize = layer->prevSize * layer->size;

		float* weights = new float[allWeightsSize];
		float* biases = new float[layer->size];

		if (i < m_hiddenLayers.size()) {
			layer = &m_hiddenLayers[i];
		}
		else {
			layer = &m_outputLayer;
		}
		for (size_t neuron = 0; neuron < layer->size; neuron++) {
			for (size_t prevNeuron = 0; prevNeuron < layer->prevSize; prevNeuron++) {
				weights[prevNeuron + neuron * layer->prevSize] = 0;
			}
			biases[neuron] = 0;
		}

		weightAcc.push_back(0);
		biasAcc.push_back(0);

		cudaStatus = hipMalloc((void**)&weightAcc.back(), allWeightsSize * sizeof(float));
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMalloc failed! weightAcc, backpropagateGPU");
			system("pause");
		}

		cudaStatus = hipMalloc((void**)&biasAcc.back(), layer->size * sizeof(float));
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMalloc failed!, biasAcc, backpropagateGPU");
			system("pause");
		}



		cudaStatus = hipMemcpy(weightAcc.back(), weights, allWeightsSize * sizeof(float), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed!, weightAcc, backpropagateGPU");
			system("pause");
		}

		cudaStatus = hipMemcpy(biasAcc.back(), biases, layer->size * sizeof(float), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed!, biasAcc, backpropagateGPU");
			system("pause");
		}

		delete[] weights;
		delete[] biases;
	}


	for (size_t i = 0; i < dCost_dOutput_forInstances.size();i++) {
		//reverse order, last layer first
		std::vector<float*> dCost_dErrorL;
		dCost_dErrorL.push_back(0);
		cudaStatus = hipMalloc((void**)&dCost_dErrorL.back(), dCost_dOutput_forInstances[i].size() * sizeof(float));
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMalloc failed!, dCost_dErrorL, backpropagateGPU %s\n", hipGetErrorString(cudaStatus));
			system("pause");
		}
		cudaStatus = hipMemcpy(dCost_dErrorL.back(), dCost_dOutput_forInstances[i].data(), dCost_dOutput_forInstances[i].size() * sizeof(float), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed!, dCost_dErrorL, backpropagateGPU");
			system("pause");
		}



		for (int o = m_hiddenLayers.size() - 1; o >= 0; o--) {
			float* nextZError = dCost_dErrorL.back();
			Layer* nextLayer = o == m_hiddenLayers.size() ? &m_outputLayer : &m_hiddenLayers[o+1];

			dCost_dErrorL.push_back(0);
			cudaStatus = hipMalloc((void**)&dCost_dErrorL.back(), m_hiddenLayers[o].size * sizeof(float));
			if (cudaStatus != hipSuccess) {
				fprintf(stderr, "hipMalloc failed!, dCost_dErrorL, loop 2, backpropagateGPU, %s\n", hipGetErrorString(cudaStatus));
				system("pause");
			}


			unsigned int blockSize = std::max((unsigned int)std::ceilf(m_hiddenLayers[o].size / THREADS_PER_BLOCK), (unsigned int)1);
			unsigned int threadSize = std::min((size_t)THREADS_PER_BLOCK, m_hiddenLayers[o].size);


			// Launch a kernel on the GPU with one thread for each element.
			//accumulate biases for this instance
			calculateZErrorKernel <<< blockSize, threadSize >>> (dCost_dErrorL.back(), biasAcc[o], m_savedZValues[i][o + 1], m_savedAValues[i][o + 1], nextZError, std::get<0>(m_GPULayers[o + 1]), std::get<2>(m_GPULayers[o]), std::get<2>(m_GPULayers[o + 1]));

			// Check for any errors launching the kernel
			cudaStatus = hipGetLastError();
			if (cudaStatus != hipSuccess) {
				fprintf(stderr, "calculateZErrorKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
				system("pause");
			}

			// hipDeviceSynchronize waits for the kernel to finish, and returns
			// any errors encountered during the launch.
			cudaStatus = hipDeviceSynchronize();
			if (cudaStatus != hipSuccess) {
				fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching calculateZErrorKernel!\n", cudaStatus);
				system("pause");
			}
			//accumulate weights for this instance
			calculateWeightGradientsKernel <<< blockSize, threadSize >>> (weightAcc[o], dCost_dErrorL.back(), m_savedAValues[i][o], std::get<2>(m_GPULayers[o]));

			// Check for any errors launching the kernel
			cudaStatus = hipGetLastError();
			if (cudaStatus != hipSuccess) {
				fprintf(stderr, "calculateWeightGradientsKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
				system("pause");
			}

			// hipDeviceSynchronize waits for the kernel to finish, and returns
			// any errors encountered during the launch.
			cudaStatus = hipDeviceSynchronize();
			if (cudaStatus != hipSuccess) {
				fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching calculateWeightGradientsKernel!\n", cudaStatus);
				system("pause");
			}
		}

		for (auto dError : dCost_dErrorL) {
			hipFree(dError);
		}

	}

	float* instancesAndLearningRate = new float[2];
	instancesAndLearningRate[0] = dCost_dOutput_forInstances.size();
	instancesAndLearningRate[1] = m_learningSched.learningRate;
	float *instancesAndLearningRateGPU=0;
	cudaStatus = hipMalloc((void**)&instancesAndLearningRateGPU, 2* sizeof(float));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
	}
	cudaStatus = hipMemcpy(instancesAndLearningRateGPU, instancesAndLearningRate, 2 * sizeof(float), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
	}


	for (size_t i = 0; i < m_hiddenLayers.size() + 1; i++) {
		const Layer* layer = (i < m_hiddenLayers.size() ? &m_hiddenLayers[i] : &m_outputLayer);
		unsigned int blockSize = std::max((unsigned int)std::ceilf(layer->size / THREADS_PER_BLOCK), (unsigned int)1);
		unsigned int threadSize = std::min((size_t)THREADS_PER_BLOCK, layer->size);
		addAverageWeightsAndBiasesKernel <<< blockSize, threadSize >>> (std::get<0>(m_GPULayers[i]), weightAcc[i], std::get<1>(m_GPULayers[i]), biasAcc[i], instancesAndLearningRateGPU, std::get<2>(m_GPULayers[i]));
		// Check for any errors launching the kernel
		cudaStatus = hipGetLastError();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "addAverageAndBiasesKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
			system("pause");
		}

		// hipDeviceSynchronize waits for the kernel to finish, and returns
		// any errors encountered during the launch.
		cudaStatus = hipDeviceSynchronize();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addAverageAndBiasesKernel!\n", cudaStatus);
			system("pause");
		}
		
	}

	hipFree(instancesAndLearningRateGPU);



	m_updateRAM();

	for (auto weight : weightAcc) {
		hipFree(weight);
	}
	for (auto bias : biasAcc) {
		hipFree(bias);
	}

	clearTrainingData();
	m_batchN++;
}

void NeuralNetwork::startRecording()
{

	m_recording = true;
}

void NeuralNetwork::endRecording()
{
	m_instanceN = 0;
	m_recording = false;

}

void NeuralNetwork::selectAndDiscardRest(unsigned int selected)
{
	m_instancesInBatch++;

	m_savedAValues.push_back(m_intermediateAValues[selected]);
	m_savedZValues.push_back(m_intermediateZValues[selected]);


	for (size_t i = 0; i < m_intermediateAValues.size(); i++) {
		if (i != selected) {
			for (auto val : m_intermediateAValues[i]) {
				hipFree(val);
			}
		}
	}
	for (size_t i = 0; i < m_intermediateZValues.size(); i++) {
		if (i != selected) {
			for (auto val : m_intermediateZValues[i]) {
				hipFree(val);
			}
		}
	}


	m_intermediateAValues.clear();
	m_intermediateZValues.clear();

}

void NeuralNetwork::clearTrainingData()
{
	for (auto& layer : m_savedAValues) {
		for (auto val : layer) {
			hipFree(val);
		}
	}
	m_savedAValues.clear();
	for (auto& layer : m_savedZValues) {
		for (auto val : layer) {
			hipFree(val);
		}
	}
	m_savedZValues.clear();
}


#include <sstream>

std::vector<float> NeuralNetwork::forwardPassCPU(const std::vector<float>& input) const
{
	if (m_top[0] != input.size()) {
		std::stringstream ss;
		ss << "input layer is not the same size as the parameters passed: " << m_top[0] << " vs " << input.size() << std::endl;
		throw std::invalid_argument(ss.str());
	}

	std::vector<float> prevResult = input;
	std::vector<float> thisResult;

	for (size_t i = 0; i < m_hiddenLayers.size() + 1; i++)
	{
		const Layer* layer;
		if (i < m_hiddenLayers.size()) {
			layer = &m_hiddenLayers[i];
		}
		else {
			layer = &m_outputLayer;
		}

		for (size_t o = 0; o < layer->size; o++) {
			auto& synapses = layer->weights[o];
			float weightedSum = 0;
			for (size_t p = 0; p < layer->prevSize;p++) {
				weightedSum += synapses[p] * prevResult[p];
			}
			thisResult.push_back(layer->act.func(weightedSum + layer->biases[o]));
		}
		prevResult = thisResult;
	}


	return thisResult;
}




__global__ void forwardPassLayerKernel(float* zValues, float* VecOut, const float* VecIn, const float* weights, const float* biases, const unsigned int* extraParams)
{
	int neuron = threadIdx.x + blockDim.x * blockIdx.x;
	if (neuron < extraParams[2]) {

		float sum = 0;
		for (size_t i = 0; i < extraParams[1]; i++) {
			sum += weights[i + neuron * extraParams[1]] * VecIn[i];
		}
		zValues[neuron] = sum+biases[neuron];
		if (extraParams[0] == 1) {
			VecOut[neuron] = sum + biases[neuron];
		}
		else if (extraParams[0] == 2) {
			float z = sum + biases[neuron];
			VecOut[neuron] = 1 / (1 + exp(-z));
		}
		else if(extraParams[0] == 3) {
			float z = sum + biases[neuron];
			VecOut[neuron] = z * (1 / (1 + exp(-z)));
		}
	}
}



void NeuralNetwork::m_updateGPUMem()
{
	// Choose which GPU to run on, change this on a multi-GPU system.
	hipError_t cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
	}


	for (size_t i = 0; i < m_hiddenLayers.size() + 1; i++) {
		const Layer* layer = (i < m_hiddenLayers.size() ? &m_hiddenLayers[i] : &m_outputLayer);

		size_t allWeightsSize = layer->prevSize * layer->size;

		float* weights = new float[allWeightsSize];
		float* biases = new float[layer->size];
		unsigned int* extraParams = new unsigned int[3];
		extraParams[0] = (unsigned int)m_act.actType;
		extraParams[1] = (unsigned int)layer->prevSize;
		extraParams[2] = (unsigned int)layer->size;

		if (i < m_hiddenLayers.size()) {
			layer = &m_hiddenLayers[i];
		}
		else {
			layer = &m_outputLayer;
		}
		for (size_t neuron = 0; neuron < layer->size; neuron++) {
			for (size_t prevNeuron = 0; prevNeuron < layer->prevSize; prevNeuron++) {
				weights[prevNeuron + neuron * layer->prevSize] = layer->weights[neuron][prevNeuron];
			}
			biases[neuron] = layer->biases[neuron];
		}
		// Copy input vectors from host memory to GPU buffers.
		cudaStatus = hipMemcpy(std::get<0>(m_GPULayers[i]), weights, allWeightsSize * sizeof(float), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed!");
		}

		cudaStatus = hipMemcpy(std::get<1>(m_GPULayers[i]), biases, layer->size * sizeof(float), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed!");
		}

		cudaStatus = hipMemcpy(std::get<2>(m_GPULayers[i]), extraParams, 3 * sizeof(unsigned int), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed! extraParams, m_updateGPUmem()");
		}
		delete[] weights;
		delete[] biases;
		delete[] extraParams;
	}

}

void NeuralNetwork::m_updateRAM()
{
	hipError_t cudaStatus = hipSetDevice(0);
	for (size_t i = 0; i < m_hiddenLayers.size() + 1; i++) {
		Layer* layer = (i < m_hiddenLayers.size() ? &m_hiddenLayers[i] : &m_outputLayer);

		size_t allWeightsSize = layer->prevSize * layer->size;

		float* weights = new float[allWeightsSize];
		float* biases = new float[layer->size];


		// Copy input vectors from host memory to GPU buffers.
		cudaStatus = hipMemcpy(weights, std::get<0>(m_GPULayers[i]), allWeightsSize * sizeof(float), hipMemcpyDeviceToHost);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed!");
		}

		cudaStatus = hipMemcpy(biases, std::get<1>(m_GPULayers[i]), layer->size * sizeof(float), hipMemcpyDeviceToHost);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed!");
		}

		if (i < m_hiddenLayers.size()) {
			layer = &m_hiddenLayers[i];
		}
		else {
			layer = &m_outputLayer;
		}
		for (size_t neuron = 0; neuron < layer->size; neuron++) {
			for (size_t prevNeuron = 0; prevNeuron < layer->prevSize; prevNeuron++) {
				layer->weights[neuron][prevNeuron] = weights[prevNeuron + neuron * layer->prevSize];
			}
			layer->biases[neuron] = biases[neuron];
		}
		delete[] weights;
		delete[] biases;
	}
}



std::vector<float> NeuralNetwork::forwardPassGPU(const std::vector<float>& input) const
{

	if (m_top[0] != input.size()) {
		std::stringstream ss;
		ss << "input layer is not the same size as the parameters passed: " << m_top[0] << " vs " << input.size() << std::endl;
		throw std::invalid_argument(ss.str());
	}


	hipError_t cudaStatus;

	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
	}

	float* inputGPU = 0;
	cudaStatus = hipMalloc((void**)&inputGPU, input.size() * sizeof(float));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
	}


	cudaStatus = hipMemcpy(inputGPU, input.data(), input.size() * sizeof(float), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
	}

	//start recording a new  instance of intermediate values, first entry in  the instance is the input.
	if (m_recording) {
		m_intermediateAValues.push_back({});
		m_intermediateZValues.push_back({});
		m_instanceN++;
		float* zValues = 0;
		cudaStatus = hipMalloc((void**)&zValues, input.size() * sizeof(float));
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMalloc failed!");
		}


		float* aValues = 0;
		cudaStatus = hipMalloc((void**)&aValues, input.size() * sizeof(float));
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMalloc failed!");
		}

		cudaStatus = hipMemcpy(zValues, input.data(), input.size() * sizeof(float), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed!");
		}

		cudaStatus = hipMemcpy(aValues, input.data(), input.size() * sizeof(float), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed!");
		}

		m_intermediateAValues.back().push_back(aValues);
		m_intermediateZValues.back().push_back(zValues);
	}



	for (size_t i = 0; i < m_hiddenLayers.size() + 1; i++) {

		const Layer* layer;
		if (i < m_hiddenLayers.size()) {
			layer = &m_hiddenLayers[i];
		}
		else {
			layer = &m_outputLayer;
		}

		float* zValues = 0;

		cudaStatus = hipMalloc((void**)&zValues, layer->size * sizeof(float));
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMalloc failed!");
		}

		float* output = 0;

		cudaStatus = hipMalloc((void**)&output, layer->size * sizeof(float));
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMalloc failed!");
		}



		unsigned int blockSize = std::max((unsigned int)std::ceilf(layer->size/THREADS_PER_BLOCK), (unsigned int)1);
		unsigned int threadSize = std::min((size_t)THREADS_PER_BLOCK,layer->size);
		forwardPassLayerKernel<<< blockSize, threadSize >>>(zValues,output, inputGPU, std::get<0>(m_GPULayers[i]), std::get<1>(m_GPULayers[i]), std::get<2>(m_GPULayers[i]));

		if (m_recording) {
			m_intermediateAValues.back().push_back(output);
			m_intermediateZValues.back().push_back(zValues);
		}
		else {
			hipFree(zValues);
		}


		// Check for any errors launching the kernel
		cudaStatus = hipGetLastError();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "processLayerKernel launch failed: %s%s\n", hipGetErrorString(cudaStatus));
			system("pause");
		}

		// hipDeviceSynchronize waits for the kernel to finish, and returns
		// any errors encountered during the launch.
		cudaStatus = hipDeviceSynchronize();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching processLayerKernel!\n", cudaStatus);
		}




		if (i == m_hiddenLayers.size()) {
			hipFree(inputGPU);
			float* out = new float[layer->size];
			// Copy output vector from GPU buffer to host memory.
			cudaStatus = hipMemcpy(out, output, layer->size * sizeof(float), hipMemcpyDeviceToHost);
			if (cudaStatus != hipSuccess) {
				fprintf(stderr, "hipMemcpy failed!");
			}
			if (!m_recording) {
				hipFree(output);
			}


			std::vector<float> retval = std::vector<float>(out, out + layer->size);
			delete out;
			return retval;
		}
		else {
			hipFree(inputGPU);

			cudaStatus = hipMalloc((void**)&inputGPU, layer->size * sizeof(float));
			if (cudaStatus != hipSuccess) {
				fprintf(stderr, "hipMalloc failed!");
			}
			cudaStatus = hipMemcpy(inputGPU, output, layer->size * sizeof(float), hipMemcpyDeviceToDevice);
			if (cudaStatus != hipSuccess) {
				fprintf(stderr, "hipMemcpy failed!");
			}
		}
		if (!m_recording) {
			hipFree(output);
		}
	}

	return std::vector<float>();
}



std::string NeuralNetwork::serialize()const
{

	std::stringstream ss;
	for (auto x : m_top)
	{
		ss << x << " ";
	}
	ss << std::endl;


	for (size_t i = 0; i < m_hiddenLayers.size()+1; i++)
	{
		const Layer *layer;
		if (i < m_hiddenLayers.size()) {
			layer = &m_hiddenLayers[i];
		}
		else {
			layer = &m_outputLayer;
		}

		for (size_t o = 0; o < layer->size; o++) {
			ss << layer->biases[o] << " ";
			auto &synapses = layer->weights[o];
			for (auto syn : synapses) {
				ss << syn << " ";
			}
			ss << "/";
		}
		ss << ",";
	}
	return ss.str();
}

NeuralNetwork::~NeuralNetwork()
{
	for (auto &GPULayer : m_GPULayers) {
		hipFree(std::get<0>(GPULayer));
		hipFree(std::get<1>(GPULayer));
		hipFree(std::get<2>(GPULayer));
	}
}

void NNInitialization::seedEngine()
{
	engine.seed();
}

float NNInitialization::generateRandomNumber(float fan)
{
	std::uniform_real_distribution<float> dist;

	float r;
	switch (initType)
	{
	case NNInitialization::He:
		r = sqrt(6 / fan);
		dist = std::uniform_real_distribution<float>(-r,r);
		break;
	default:
		r = 1;
		dist = std::uniform_real_distribution<float>(-r, r);
		break;
	}

	return dist(NNInitialization::engine);
}


void Layer::randomizeParameters(LearningSchedule& ls)
{
	for (size_t i = 0; i < size; i++) {
		weights.push_back(std::vector<float>());
		for (size_t o = 0; o < prevSize; o++) {
			weights[i][o] += ls.generateRandomNumber();
		}
	}
	if (!isOutput) {
		for (size_t i = 0; i < size; i++) {
			biases[i] += ls.generateRandomNumber();
		}
	}
}

Layer::Layer(NNInitialization& init, size_t p_size, size_t p_prevSize, bool p_isOutput, NNActivation p_act):
	size(p_size),
	prevSize(p_prevSize),
	act(p_act),
	isOutput(p_isOutput)
{

	for (size_t i = 0; i < size; i++) {
		weights.push_back(std::vector<float>());
		for (size_t o = 0; o < prevSize; o++) {
			weights[i].push_back(init.generateRandomNumber(p_prevSize));
		}
	}
	if (isOutput) {
		biases = std::vector<float>(size, 0.0f);
		act = NNActivation(activationType::linear);
	}
	else {
		for (size_t i = 0; i < size; i++) {
			biases.push_back(init.generateRandomNumber(p_prevSize));
		}
	}
}

Layer::Layer():
	act(NNActivation(activationType::sigmoid))
{
}

NNActivation::NNActivation(activationType p_actType)
{
	if (p_actType == activationType::other) {
		throw std::invalid_argument("Must be activation type that is not other");
	}
	actType = p_actType;
	switch (p_actType) {
	case activationType::sigmoid:
		func = [](float in) {return 1 / (1 + exp(-in));};
		break;
	case activationType::linear:
		func = [](float in) {return in;};
		break;
	case activationType::swish:
		func = [](float in) {return in*(1 / (1 + exp(-in)));};
	}
}

NNActivation::NNActivation(activationFunc p_func)
{
	actType = activationType::other;
	func = p_func;
}


float LearningSchedule::generateRandomNumber()
{
	return dist(NNInitialization::engine);
}
